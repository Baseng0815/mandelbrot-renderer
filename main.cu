#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stddef.h>
#include <math.h>

#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

struct comp {
        double real;
        double imag;
};

const int max_iter          = 2048;
const int img_width         = 1920;
const int img_height        = 1080;
const struct comp center    = {
        .real = -0.77568377,
        .imag = 0.13646737
};

const double zoom_start     = 1.0;
const double zoom_end       = 1.0e227;
const double zoom_fact      = 1.1;

__device__ void iterate(struct comp c, int *iterations)
{
        struct comp z = c;
        int i = 0;
        while (i < max_iter &&
               sqrt(z.real * z.real + z.imag * z.imag) < 2.0) {
                double real = z.real * z.real - z.imag * z.imag + c.real;
                double imag = z.real * z.imag * 2.0 + c.imag;
                z.real = real;
                z.imag = imag;
                i++;
        }

        *iterations = i;
}

__device__ void map_pixel(int px, int py, double zoom,
                          struct comp center, struct comp *result)
{
        double span_real    = 4.0 / zoom;
        double span_imag    = 2.0 / zoom;

        /* first without taking zoom into account, but already centered */
        double real_nozoom = span_real / (double)img_width  * px
                - span_real / 2.0 + center.real;
        double imag_nozoom = span_imag / (double)img_height * py
                - span_imag / 2.0 + center.imag;

        result->real = real_nozoom;
        result->imag = imag_nozoom;
}

__device__ void hsv_to_rgb(int h, int s, int v, int *r, int *g, int *b)
{
        double S = s / 100.0;
        double V = v / 100.0;
        double c = S * V;
        double x = c * (1.0 - fabs(fmod(h / 60.0, 2.0) - 1.0));
        double m = V - c;

        double R, G, B;

        if (h >= 0 && h < 60)
                R = c, G = x, B = 0;
        else if (h >= 60 && h < 180)
                R = x, G = c, B = x;
        else if (h >= 180 && h < 240)
                R = 0, G = x, B = c;
        else if (h >= 240 && h < 300)
                R = x, G = 0, B = c;
        else
                R = c, G = 0, B = x;

        *r = (R + m) * 255.0;
        *g = (G + m) * 255.0;
        *b = (B + m) * 255.0;
}

__global__ void plot_pixel(uint8_t *buf, struct comp center, double zoom)
{
        /* get index in grid-stride loop and convert to pixel coordinates */
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int px = i % img_width;
        int py = i / img_width;

        struct comp c;
        map_pixel(px, py, zoom, center, &c);

        int iterations;
        iterate(c, &iterations);

        int r, g, b;
        hsv_to_rgb(iterations % 360, 100,
                   (max_iter - iterations) /
                   (double)max_iter * 100, &r, &g, &b);

        size_t byte = 3 * (py * img_width + px);
        buf[byte + 0] = r;
        buf[byte + 1] = g;
        buf[byte + 2] = b;
}

void plot_mandelbrot(uint8_t *buf, double zoom)
{
        /* this should be a multiple of img_width * img_height */
        int threads_per_block   = 256;
        int block_count         = img_width * img_height / threads_per_block;

        plot_pixel<<<block_count, threads_per_block>>>(buf, center, zoom);
        hipDeviceSynchronize();
}

int main(int argc, char **argv)
{
        struct stat st = { 0 };
        if (stat("./images", &st) == -1) {
                mkdir("./images", 0755);
        }

        uint8_t *buf;
        hipMallocManaged(&buf, img_width * img_height * 3);

        int img_id = 0;
        int img_count = log(zoom_end / zoom_start) / log(zoom_fact);
        printf("Total number of images to plot: %d\n", img_count);
        for (double zoom = zoom_start; zoom <= zoom_end; zoom *= zoom_fact) {
                printf("Plotting image %d with zoom %f (%d%)...\n",
                       img_id, zoom, img_id * 100 / img_count);
                plot_mandelbrot(buf, zoom);

                char file_name[32];
                sprintf(file_name, "images/%06d.jpeg", img_id);
                stbi_write_jpg(file_name, img_width, img_height, 3, buf, 100);

                img_id++;
        }

        hipFree(buf);
}
